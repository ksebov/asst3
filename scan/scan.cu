#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

inline static int ceilLog2(int32_t val) {
  assert(val > 0);

  int n = 0;
  while (val > (1 << n)) {
    n++;
  }

  return n;
}

void exclusive_scan_iterative(int logN, int* output) {
  // upsweep phase
  for (int d = 0; d < logN-2; ++d) {
    /*parallel*/for (int k = 0; k < (1 << (logN - d - 1)); ++k) {
      const int ix0 = (k << (d + 1)) + (1 << d) - 1;
      const int ix1 = ix0 + (1 << d);

      output[ix1] += output[ix0];
    }
  }

  // junction phase
  const int ix0 = (1 << (logN - 2)) - 1;
  const int ix1 = (1 << (logN - 1)) - 1;

  output[(1<<logN) - 1] = output[ix0] + output[ix1];
  output[ix1] = 0;

  // downsweep phase
  for (int d = logN-2; d >= 0; --d) {
    /*parallel*/for (int k = 0; k < (1 << (logN - d - 1)); ++k) {
      const int ix0 = (k << (d + 1)) + (1 << d) - 1;
      const int ix1 = ix0 + (1 << d);

      const int t = output[ix0];
      output[ix0] = output[ix1];
      output[ix1] += t;
    }
  }
}

__global__ void upsweep_kernel(int d, int* output) {
  const int k = blockIdx.x * blockDim.x + threadIdx.x;

  const int ix0 = (k << (d + 1)) + (1 << d) - 1;
  const int ix1 = ix0 + (1 << d);

  output[ix1] += output[ix0];
}

__global__ void junction_kernel(int* output, int logN) {
  const int ix0 = (1 << (logN - 2)) - 1;
  const int ix1 = (1 << (logN - 1)) - 1;

  output[(1 << logN) - 1] = output[ix0] + output[ix1];
  output[ix1] = 0;
}

__global__ void downsweep_kernel(int d, int* output) {
  const int k = blockIdx.x * blockDim.x + threadIdx.x;

  const int ix0 = (k << (d + 1)) + (1 << d) - 1;
  const int ix1 = ix0 + (1 << d);

  const int t = output[ix0];
  output[ix0] = output[ix1];
  output[ix1] += t;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel segmented scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
static void exclusive_scan(int logN, int* output) {
  for (int d = 0; d < logN - 2; ++d) {
    const int threads = 1 << (logN - d - 1);
    const int threadsPerBlock = std::min(threads, THREADS_PER_BLOCK);

    upsweep_kernel<<< threads/threadsPerBlock, threadsPerBlock >>> (d, output);
  }

  junction_kernel <<< 1,1 >>> (output, logN);

  for (int d = logN - 2; d >= 0; --d) {
    const int threads = 1 << (logN - d - 1);
    const int threadsPerBlock = std::min(threads, THREADS_PER_BLOCK);

    downsweep_kernel << < threads / threadsPerBlock, threadsPerBlock >> > (d, output);
  }
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of segmented scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    const int N = end - inarray;
    const int logN = ceilLog2(N);
# if 0
    {
      assert(N == (1 << logN));
      memmove(resultarray, inarray, N * sizeof(int));

      const double startTime = CycleTimer::currentSeconds();
      exclusive_scan_iterative(logN, resultarray);
      const double endTime = CycleTimer::currentSeconds();

      return endTime - startTime;
    }
#endif
    int* device_result;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    hipMalloc((void **)&device_result, sizeof(int) << logN);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_result, inarray, N * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(logN, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, N * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    return 0; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
